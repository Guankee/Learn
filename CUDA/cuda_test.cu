#include <chrono>
#include <iostream>
#include <thread>

#include "hip/hip_runtime.h"

#define N (50)
__global__ void add(int* a, int* b, int* c) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  while (tid < N) {
    clock_t start_time = clock();
    while ((clock() - start_time) < 100) {
    }
    c[tid] = a[tid] + b[tid];

    tid += gridDim.x + blockDim.x;
  }
}
int main() {
  auto st = std::chrono::system_clock::now();
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;
  hipMalloc((void**)&dev_a, N * sizeof(int));
  hipMalloc((void**)&dev_b, N * sizeof(int));
  hipMalloc((void**)&dev_c, N * sizeof(int));
  for (int i = 0; i < N; ++i) {
    a[i] = -i;
    b[i] = i * i;
  }
  hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  add<<<128, 128>>>(dev_a, dev_b, dev_c);
  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    std::cout << c[i] << " ";
  }
  std::cout << std::endl;
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  auto end = std::chrono::system_clock::now();
  std::cout
      << std::chrono::duration_cast<std::chrono::milliseconds>(end - st).count()
      << "ms" << std::endl;
  return 0;
}